#include "hip/hip_runtime.h"
#include "SPHSolver.cuh"
#include "SPHKernels.cuh"
#include "stdio.h"
#include "types.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <climits>
#include <glm/gtx/color_space.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 256

__constant__ glm::ivec3 NEIGH_DISPLACEMENTS[27];
#define forall_fluid_neighbors(code)\
    for (int neighDispIdx = 0; neighDispIdx < 27; ++neighDispIdx)\
    {\
        glm::ivec3 neighborIndex = cellIdx + NEIGH_DISPLACEMENTS[neighDispIdx];\
        uint32_t neighborHashedCell = getHashedCell(neighborIndex, size * 2);\
        uint32_t neighborIterator = cellOffsetBuffer[neighborHashedCell];\
        while(neighborIterator != (size * 2) && neighborIterator < size)\
        {\
            uint32_t j = particleIndexBuffer[neighborIterator];\
            if(cellIndexBuffer[j] != neighborHashedCell)\
            {\
                break;\
            }\
            code\
            neighborIterator++;\
        }\
    }\

// (80000 particulas y + 50 fps)
__device__ uint32_t getHashedCell2(glm::ivec3 cellIdx, uint32_t size)
{
    const uint32_t p1 = 73856093; 
    const uint32_t p2 = 19349663;
    const uint32_t p3 = 83492791;

    int n = p1 * cellIdx.x ^ p2 *cellIdx.y ^ p3 * cellIdx.z;
    n %= size;

    return n;
}

// No negativos (con 80000 particulas 71 fps)
__device__ uint32_t getHashedCell1(const glm::ivec3 key, const uint32_t size) 
{
    uint32_t hash = 2166136261u;
    const uint32_t prime = 16777619u;
    const uint32_t x = key.x;
    const uint32_t y = key.y;
    const uint32_t z = key.z;
    
    hash = (hash ^ x) * prime;
    hash = (hash ^ y) * prime;
    hash = (hash ^ z) * prime;
    
    return hash % size;
}

// Admite negativos (con 80000 particulas ~ 79 fps)
__device__ uint32_t getHashedCell(const glm::ivec3 key, const uint32_t size) 
{
    const uint32_t seed = 0xDEADBEEF;
    uint32_t hash = seed;
    const uint32_t prime = 16777619u;
    const uint32_t x = static_cast<uint32_t>(key.x);
    const uint32_t y = static_cast<uint32_t>(key.y);
    const uint32_t z = static_cast<uint32_t>(key.z);
    
    hash ^= x + seed;
    hash *= prime;
    hash ^= y + seed;
    hash *= prime;
    hash ^= z + seed;
    hash *= prime;
    
    return hash % size;
}

__global__ void computeDensityHashed(glm::vec4* positions, float* densities, float* pressures, uint32_t* particleIndexBuffer, uint32_t* cellIndexBuffer, uint32_t* cellOffsetBuffer, float* h_ptr, float* mass_ptr, float* density0_ptr, float* cubicConst_ptr, float* stiffness_ptr, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        float h = *h_ptr;
        float mass = *mass_ptr;
        float cubicConst = *cubicConst_ptr;
        float density0 = *density0_ptr;
        float stiffness = *stiffness_ptr;
        glm::vec4 ri = positions[i];
        glm::vec3 pos = glm::vec3(ri);

        float density = 0.0f;
        glm::ivec3 cellIdx = glm::floor(pos / h);

        forall_fluid_neighbors
        (
            glm::vec3 rij = glm::vec3(ri - positions[j]);
            density += cubicW(rij, h, cubicConst);
        );

        density *= mass;
        densities[i] = density;
        pressures[i] = max(0.0f, stiffness * (density - density0));
    }
}

__global__ void computePressureForceCudaHashed(glm::vec4* positions, glm::vec3* forces, glm::vec3* velocities, float* densities, float* pressures, uint32_t* particleIndexBuffer, uint32_t* cellIndexBuffer, uint32_t* cellOffsetBuffer, float* h_ptr, float* mass_ptr, float* spikyConst_ptr, float* viscosity_ptr, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        float h = *h_ptr;
        float mass = *mass_ptr;
        float spikyConst = *spikyConst_ptr;
        float visco = *viscosity_ptr;
        float pi = pressures[i];
        float di = densities[i];
        glm::vec4 ri = positions[i];
        glm::vec3 vi = velocities[i];

        glm::vec3 pforce = glm::vec3(0.0f);
        glm::vec3 vforce = glm::vec3(0.0f);
        glm::vec3 pos = glm::vec3(ri);
        glm::ivec3 cellIdx = glm::floor(pos / h);

        forall_fluid_neighbors
        (
            glm::vec3 rij = glm::vec3(ri - positions[j]);

            pforce -= (pressures[j] / (densities[j] * densities[j]) + pi / (di * di)) * spikyW(rij, h, spikyConst);
            vforce += (velocities[j] - vi) / densities[j] * laplW(rij, h, spikyConst);
        );

        forces[i] += (vforce * visco + pforce) * mass;
    }
}

__global__ void integrationCuda(glm::vec4* positions, glm::vec3* forces, glm::vec3* velocities, glm::vec4* colors, float* timeStep_ptr, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        float timeStep = *timeStep_ptr;
        glm::vec3 G = glm::vec3(0, -9.8f, 0);

        velocities[i] += timeStep * (forces[i] + G);
        positions[i] += glm::vec4(timeStep * velocities[i], 0.0f);

        forces[i] = glm::vec3(0.0f);

        // Coloreado con velocidad
        float speed = glm::length(velocities[i]);
        const float maxSpeed = 4.0f; 
        glm::vec3 hsvMin = glm::vec3(210.0f, 1.0f, 1.0f); 
        glm::vec3 hsvMax = glm::vec3(210.0f, 0.13f, 1.0);
        glm::vec3 hsv = glm::mix(hsvMin, hsvMax, speed / maxSpeed);
        colors[i] = glm::vec4(glm::rgbColor(hsv), 1.0f);
    }
}

__global__ void simpleBoundaryConditionCuda(glm::vec4* positions, glm::vec3* velocities, glm::vec3* min_ptr, glm::vec3* max_ptr, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        glm::vec3 pos = glm::vec3(positions[i]);
        glm::vec3 vel = glm::vec3(velocities[i]);
        glm::vec3 min = *min_ptr;
        glm::vec3 max = *max_ptr;
        float restitution = 0.01f;

        if (pos.x < min.x)
        {
            pos.x = min.x;
            vel.x *= - restitution;
        }
        else if (pos.x > max.x)
        {
            pos.x = max.x;
            vel.x *= - restitution;
        }

        if (pos.y < min.y)
        {
            pos.y = min.y;
            vel.y *= - restitution;
        }
        else if (pos.y > max.y)
        {
            pos.y = max.y;
            vel.y *= - restitution;
        }

        if (pos.z < min.z)
        {
            pos.z = min.z;
            vel.z *= - restitution;
        }
        else if (pos.z > max.z)
        {
            pos.z = max.z;
            vel.z *= - restitution;
        }

        positions[i] = glm::vec4(pos, 1.0f);
        velocities[i] = vel;
    }
}

__global__ void resetCuda(glm::vec3* velocities, glm::vec3* forces, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        velocities[i] = glm::vec3(0.0f);
        forces[i] = glm::vec3(0.0f);
    }
}

__global__ void resetOffset(uint32_t* cellOffset, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr * 2;

    if (i < size)
    {
        cellOffset[i] = size;
    }
}

__global__ void resetparticleIndexBuffer(uint32_t* particleIndexBuffer, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        particleIndexBuffer[i] = i; 
    }
}

__global__ void insertParticles(glm::vec4* positions, uint32_t* particleIndexBuffer, uint32_t* cellIndexBuffer, glm::vec4* colors, float* h_ptr, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        float h = *h_ptr;
        uint32_t idx = particleIndexBuffer[i];
        glm::vec3 pos = glm::vec3(positions[idx]);

        glm::ivec3 cell = glm::floor(pos / h);
        uint32_t hashedCell = getHashedCell(cell, size * 2);
        cellIndexBuffer[idx] = hashedCell;

        // Color por celda
        // hiprandState_t state;
        // hiprand_init(hashedCell, 0, 0, &state); 
        // glm::vec4 randColor = glm::vec4(
        //     hiprand_uniform(&state),
        //     hiprand_uniform(&state),
        //     hiprand_uniform(&state),
        //     1.0f
        // );
        // colors[idx] = randColor;

    }
}

__global__ void computeCellOffset(uint32_t* particleIndexBuffer, uint32_t* cellIndexBuffer, uint32_t* cellOffsetBuffer, int* size_ptr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int size = *size_ptr;

    if (i < size)
    {
        uint32_t hashedCell = cellIndexBuffer[particleIndexBuffer[i]];
        atomicMin(&cellOffsetBuffer[hashedCell], uint32_t(i));
    }
}


void SPHSolver::step(VAO_t vao)
{
    size_t bytes;
    glm::vec4* d_positions;
    glm::vec4* d_colors;

    // Map resources (positions and colors)
    gpuErrchk(hipGraphicsMapResources(1, &vao.cuda_p_id, 0)); 
    gpuErrchk(hipGraphicsMapResources(1, &vao.cuda_c_id, 0)); 
    // Get pointers of mapped data (positions and colors)
    gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&d_positions, &bytes, vao.cuda_p_id)); 
    gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&d_colors, &bytes, vao.cuda_c_id)); 

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((unsigned) ceil(2 * *size / (float) BLOCK_SIZE));

    // Reset celloffsets
    resetOffset<<<gridDim, blockDim>>>(d_cellOffsetBuffer, d_size);

    gridDim = dim3((unsigned) ceil(*size / (float) BLOCK_SIZE));

    // Reset particle index buffer
    resetparticleIndexBuffer<<<gridDim, blockDim>>>(d_particleIndexBuffer, d_size);
    
    // Insert Particles 
    insertParticles<<<gridDim, blockDim>>>(d_positions, d_particleIndexBuffer, d_cellIndexBuffer, d_colors, d_h, d_size);

    // Sort
    thrust::device_ptr<uint32_t> particleIndexPtr = thrust::device_pointer_cast(d_particleIndexBuffer);
    thrust::device_ptr<uint32_t> cellIndexPtr = thrust::device_pointer_cast(d_cellIndexBuffer);
    thrust::sort(particleIndexPtr, particleIndexPtr + *size, [cellIndexPtr] __device__ (int a, int b) { 
        return cellIndexPtr[a] < cellIndexPtr[b]; 
    });

    // Cell offsets
    computeCellOffset<<<gridDim, blockDim>>>(d_particleIndexBuffer, d_cellIndexBuffer, d_cellOffsetBuffer, d_size);

    computeDensityHashed<<<gridDim, blockDim>>>(d_positions, d_densities, d_pressures, d_particleIndexBuffer, d_cellIndexBuffer, d_cellOffsetBuffer, d_h, d_mass, d_density0, d_cubicConstK, d_stiffness, d_size);
    gpuErrchk(hipGetLastError());

    computePressureForceCudaHashed<<<gridDim, blockDim>>>(d_positions, d_forces, d_velocities, d_densities, d_pressures, d_particleIndexBuffer, d_cellIndexBuffer, d_cellOffsetBuffer, d_h, d_mass, d_spikyConst, d_viscosity, d_size);
    gpuErrchk(hipGetLastError());

    integrationCuda<<<gridDim, blockDim>>>(d_positions, d_forces, d_velocities, d_colors, d_timeStep, d_size);
    gpuErrchk(hipGetLastError());

    simpleBoundaryConditionCuda<<<gridDim, blockDim>>>(d_positions, d_velocities, d_minDomain, d_maxDomain, d_size);
    gpuErrchk(hipGetLastError());

    // Unmap resources
    gpuErrchk(hipGraphicsUnmapResources(1, &vao.cuda_p_id, 0)); 
    gpuErrchk(hipGraphicsUnmapResources(1, &vao.cuda_c_id, 0)); 
}

void SPHSolver::init()
{
    float h = *this->h;
    int size = *this->size;

    // Variables que no existen fuera de la clase SPHSolver y por lo tanto se reserva memoria para ellas
    this->spikyConst = new float(45.0f / ((float) (M_PI * pow(h, 6.0))));
    this->cubicConstK = new float(8.0f / ((float) M_PI * h * h * h));   
    this->particleIndexBuffer = new uint32_t[size];
    this->cellIndexBuffer = new uint32_t[size];
    this->cellOffsetBuffer = new uint32_t[2 * size];

    for (int i = 0; i < size; ++i)
    {
        particleIndexBuffer[i] = (uint32_t) i;
    }

    allocateCudaMemory();
}

void SPHSolver::reset(hipGraphicsResource* positionBufferObject, glm::vec4* h_positions)
{
    size_t bytes;
    glm::vec4* d_positions;

    gpuErrchk(hipGraphicsMapResources(1, &positionBufferObject, 0)); // Map resources
    gpuErrchk(hipGraphicsResourceGetMappedPointer((void**)&d_positions, &bytes, positionBufferObject)); // Get pointer of mapped data

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((unsigned) ceil(*size / (float) BLOCK_SIZE));
    resetCuda<<<gridDim, blockDim>>>(d_velocities, d_forces, d_size);
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipGraphicsUnmapResources(1, &positionBufferObject, 0)); // Unmap resources

    // Copy positions
    gpuErrchk(hipMemcpy(d_positions, h_positions, sizeof(glm::vec4) * *size, hipMemcpyHostToDevice));
}

void SPHSolver::allocateCudaMemory()
{
    // Reservar memoria en GPU
    gpuErrchk(hipMalloc(&d_h, sizeof(float)));
    gpuErrchk(hipMalloc(&d_timeStep, sizeof(float)));
    gpuErrchk(hipMalloc(&d_cubicConstK, sizeof(float)));
    gpuErrchk(hipMalloc(&d_spikyConst, sizeof(float)));
    gpuErrchk(hipMalloc(&d_radius, sizeof(float)));
    gpuErrchk(hipMalloc(&d_mass, sizeof(float)));
    gpuErrchk(hipMalloc(&d_density0, sizeof(float)));
    gpuErrchk(hipMalloc(&d_stiffness, sizeof(float)));
    gpuErrchk(hipMalloc(&d_viscosity, sizeof(float)));
    gpuErrchk(hipMalloc(&d_size, sizeof(int)));
    gpuErrchk(hipMalloc(&d_densities, sizeof(float) * *size));
    gpuErrchk(hipMalloc(&d_pressures, sizeof(float) * *size));
    gpuErrchk(hipMalloc(&d_forces, sizeof(glm::vec3) * *size));
    gpuErrchk(hipMalloc(&d_velocities, sizeof(glm::vec3) * *size));
    gpuErrchk(hipMalloc(&d_minDomain, sizeof(glm::vec3)));
    gpuErrchk(hipMalloc(&d_maxDomain, sizeof(glm::vec3)));
    gpuErrchk(hipMalloc(&d_cellIndexBuffer, sizeof(uint32_t) * (*size)));
    gpuErrchk(hipMalloc(&d_particleIndexBuffer, sizeof(uint32_t) * (*size)));
    gpuErrchk(hipMalloc(&d_cellOffsetBuffer, sizeof(uint32_t) * (*size * 2)));

    // Copiar datos a GPU
    gpuErrchk(hipMemcpy(d_h, h, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_timeStep, timeStep, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cubicConstK, cubicConstK, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_spikyConst, spikyConst, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_radius, radius, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mass, mass, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_density0, density0, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_stiffness, stiffness, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_viscosity, viscosity, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_size, size, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_densities, densities, sizeof(float) * (*size), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_pressures, pressures, sizeof(float) * (*size), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_forces, forces, sizeof(float) * (*size), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_velocities, velocities, sizeof(float) * (*size), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_minDomain, minDomain, sizeof(glm::vec3), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_maxDomain, maxDomain, sizeof(glm::vec3), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cellIndexBuffer, cellIndexBuffer, sizeof(uint32_t) * (*size), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_particleIndexBuffer, particleIndexBuffer, sizeof(uint32_t) * (*size), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cellOffsetBuffer, cellOffsetBuffer, sizeof(uint32_t) * (*size * 2), hipMemcpyHostToDevice));

    // Copy constant symbol
    glm::ivec3 displacements[] = { glm::ivec3(-1, -1, -1), glm::ivec3(-1, -1, 0), glm::ivec3(-1, -1, 1), glm::ivec3(-1, 0, -1), glm::ivec3(-1, 0, 0), glm::ivec3(-1, 0, 1), glm::ivec3(-1, 1, -1), glm::ivec3(-1, 1, 0), glm::ivec3(-1, 1, 1), glm::ivec3(0, -1, -1), glm::ivec3(0, -1, 0), glm::ivec3(0, -1, 1), glm::ivec3(0, 0, -1), glm::ivec3(0, 0, 0), glm::ivec3(0, 0, 1), glm::ivec3(0, 1, -1), glm::ivec3(0, 1, 0), glm::ivec3(0, 1, 1), glm::ivec3(1, -1, -1), glm::ivec3(1, -1, 0), glm::ivec3(1, -1, 1), glm::ivec3(1, 0, -1), glm::ivec3(1, 0, 0), glm::ivec3(1, 0, 1), glm::ivec3(1, 1, -1), glm::ivec3(1, 1, 0), glm::ivec3(1, 1, 1)};
    hipMemcpyToSymbol(HIP_SYMBOL(NEIGH_DISPLACEMENTS), &displacements[0], sizeof(glm::ivec3) * 27);
}

void SPHSolver::freeCudaMemory()
{
    gpuErrchk(hipFree(d_h));
    gpuErrchk(hipFree(d_cubicConstK));
    gpuErrchk(hipFree(d_spikyConst));
    gpuErrchk(hipFree(d_radius));
    gpuErrchk(hipFree(d_mass));
    gpuErrchk(hipFree(d_stiffness));
    gpuErrchk(hipFree(d_viscosity));
    gpuErrchk(hipFree(d_size));
    gpuErrchk(hipFree(d_densities));
    gpuErrchk(hipFree(d_pressures));
    gpuErrchk(hipFree(d_forces));
    gpuErrchk(hipFree(d_velocities));
    gpuErrchk(hipFree(d_minDomain));
    gpuErrchk(hipFree(d_maxDomain));
    gpuErrchk(hipFree(d_cellIndexBuffer));
    gpuErrchk(hipFree(d_particleIndexBuffer));
    gpuErrchk(hipFree(d_cellOffsetBuffer));
}

void SPHSolver::release()
{
    freeCudaMemory();

    delete spikyConst;
    delete cubicConstK;
    delete[] particleIndexBuffer;
    delete[] cellIndexBuffer;
    delete[] cellOffsetBuffer;
}

